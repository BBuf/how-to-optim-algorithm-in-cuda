#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

template <typename T>
void gen_rand_data(T *data, int n);

template <typename T, int kTileM, int kTileN, int kTileK, typename TiledMMA>
__global__ void gemm_simple(T *Cptr, const T *Aptr, const T *Bptr, int m, int n, int k) {

  using namespace cute;

  Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(m, k), make_stride(k, Int<1>{}));
  Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
  Tensor C = make_tensor(make_gmem_ptr(Cptr), make_shape(m, n), make_stride(n, Int<1>{}));

  int ix = blockIdx.x;
  int iy = blockIdx.y;

  Tensor gA = local_tile(A, make_tile(Int<kTileM>{}, Int<kTileK>{}), make_coord(iy, _));
  Tensor gB = local_tile(B, make_tile(Int<kTileN>{}, Int<kTileK>{}), make_coord(ix, _));
  Tensor gC = local_tile(C, make_tile(Int<kTileM>{}, Int<kTileN>{}), make_coord(iy, ix));
  //  gA(kTileM, kTileK, num_tile_k)
  //  gB(kTileN, kTileK, num_tile_k)
  //  gC(kTileM, kTileN) 

  TiledMMA tiled_mma;
  auto thr_mma = tiled_mma.get_slice(threadIdx.x);
  auto tAgA = thr_mma.partition_A(gA);  // (MMA, MMA_M, MMA_K, num_tile_k)
  auto tBgB = thr_mma.partition_B(gB);  // (MMA, MMA_N, MMA_K, num_tile_k)
  auto tCgC = thr_mma.partition_C(gC);  // (MMA, MMA_M, MMA_N)

  auto tArA = thr_mma.partition_fragment_A(gA(_, _, 0));  // (MMA, MMA_M, MMA_K)
  auto tBrB = thr_mma.partition_fragment_B(gB(_, _, 0));  // (MMA, MMA_N, MMA_K)
  auto tCrC = thr_mma.partition_fragment_C(gC(_, _));     // (MMA, MMA_M, MMA_N)
 
  clear(tCrC);
  
  int num_tile_k = size<2>(gA);
#pragma unroll 1
  for(int itile = 0; itile < num_tile_k; ++itile) {
    cute::copy(tAgA(_, _, _, itile), tArA);
    cute::copy(tBgB(_, _, _, itile), tBrB);

    cute::gemm(tiled_mma, tCrC, tArA, tBrB, tCrC);
  }

  cute::copy(tCrC, tCgC); 
}

int main() {
  srand(10086);

  using T = cute::half_t;
  using namespace cute;

  T *Cptr;
  T *Aptr;
  T *Bptr;

  int m = 81920;
  int n = 256;
  int k = 256;

  hipMalloc(&Cptr, sizeof(T) * m * n);
  hipMalloc(&Aptr, sizeof(T) * m * k);
  hipMalloc(&Bptr, sizeof(T) * k * n);

  T *Aptr_host;
  T *Bptr_host;
  Aptr_host = (T*)malloc(sizeof(T) * m * k);
  Bptr_host = (T*)malloc(sizeof(T) * n * k);
  gen_rand_data(Aptr_host, m * k);
  gen_rand_data(Bptr_host, n * k);

  hipMemcpy(Aptr, Aptr_host, sizeof(T) * m * k, hipMemcpyHostToDevice);
  hipMemcpy(Bptr, Bptr_host, sizeof(T) * n * k, hipMemcpyHostToDevice);

  using mma_op = SM80_16x8x16_F16F16F16F16_TN;
  using mma_traits = MMA_Traits<mma_op>;
  using mma_atom = MMA_Atom<mma_traits>;

  using MMA = decltype(make_tiled_mma(mma_atom{}, 
                      make_layout(Shape<_2, _2, _1>{}), 
                      make_layout(Shape<_1, _2, _1>{})));
  constexpr int kTileM = 128; 
  constexpr int kTileN = 128; 
  constexpr int kTileK = 32; 

  dim3 block(size(MMA{}));
  dim3 grid(n / kTileN, m / kTileM);
  for (int i = 0; i < 100; ++i) {
    gemm_simple<T, kTileM, kTileN, kTileK, MMA><<<grid, block>>>(Cptr, Aptr, Bptr, m, n, k);
  }
  hipDeviceSynchronize();
  auto err = hipGetLastError();
  printf("err = %d, str = %s\n", err, hipGetErrorString(err));

  // cublas
  T *Cptr_cublas;

  hipMalloc(&Cptr_cublas, sizeof(T) * m * n);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  half alpha = half(1.f);
  half beta = half(0.f);
  for (int i = 0; i < 100; ++i) {
    hipblasStatus_t ret = hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
          	  n, m, k,
          	  &alpha,
          	  (half *)Bptr, k,
          	  (half *)Aptr, k,
          	  &beta,
          	  (half *)Cptr_cublas, n);
    if (ret != HIPBLAS_STATUS_SUCCESS) {
      printf("blas err = %d, str = %s\n", ret, cublasGetStatusString(ret));
    }
  }

  hipDeviceSynchronize();
  err = hipGetLastError();
  printf("err = %d, str = %s\n", err, hipGetErrorString(err));

  T *Cptr_host;
  T *Cptr_cublas_host;

  Cptr_host = (T*)malloc(sizeof(T) * m * n);
  Cptr_cublas_host = (T*)malloc(sizeof(T) * m * n);

  // compare
  hipMemcpy(Cptr_host, Cptr, sizeof(T) * m * n, hipMemcpyDeviceToHost);
  hipMemcpy(Cptr_cublas_host, Cptr_cublas, sizeof(T) * m * n, hipMemcpyDeviceToHost);

  float threshold = 0.1;
  for (int i = 0; i < m * n; ++i) {
    float v1 = Cptr_host[i];
    float v2 = Cptr_cublas_host[i];
    if (fabs(v2 - v1) > threshold) {
      printf("v1 = %f, v2 = %f\n", v1, v2);
    }
  }

  Tensor tensor_C = make_tensor(Cptr_host, make_shape(m, n), make_stride(n, 1));
  Tensor tensor_C_cublas = make_tensor(Cptr_cublas_host, make_shape(m, n), make_stride(n, 1));

  auto tile = make_tile(8, 8);
  auto coor = make_coord(0, 0);
  Tensor tc1 = local_tile(tensor_C, tile, coor);
  Tensor tc1_cublas = local_tile(tensor_C_cublas, tile, coor);

  print_tensor(tc1);
  print_tensor(tc1_cublas);
}

template <typename T>
void gen_rand_data(T *data, int n) {
  for (int i = 0; i < n; ++i) {
    float v = (rand() % 200 - 100) * 0.01;
    data[i] = v;
  }
}
